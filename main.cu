
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <fstream>

using namespace std;
__global__ 
void AverageFinder(int* dM, int *dQ, int d_rows, int d_cols, int q_rows, int q_cols, int qavg, int th1)
{
	int avg = 0;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	// printf("threadidx:%d\n",i);
	for(int r = 0; r<q_rows; r++){
		for(int c = 0; c<q_cols; c++){
			avg += (dM[i*3 + r*d_cols*3 + c*3] + dM[i*3 + r*d_cols*3 + c*3 + 1] + dM[i*3 + r*d_cols*3 + c*3 + 2])/3;
		}
	}

	avg /= (q_rows * q_cols);
	//printf("avg : %d\n",avg);
	if(abs(qavg - avg) <= th1){
		double total = 0;
		for(int r = 0; r<q_rows; r++){
			for(int c = 0; c<q_cols; c++){
				for(int k = 0; k<3; k++){
					long v = dM[i*3 + r*d_cols*3 + c*3 + k] - dQ[r*q_cols*3 + c*3 + k];
					total += v * v;
				}
			}
		}
		total /= (q_cols*q_rows*3);
		total = sqrt(total);
		printf("%d is close, RMSD : %f\n",i,total);
	}

}

int main(int argc, char* argv[]){
	if(argc < 5){
		cout<<"insufficient args provided\n";
		return -1;
	}

	ifstream image_file(argv[1], ios::in);
	ifstream query_file(argv[2], ios::in);
	int threshold = atoi(argv[3]);
	int topn = atoi(argv[4]);

	int d_rows,d_cols;
	image_file>>d_rows;
	image_file>>d_cols;

	int *input_img = new int[d_rows * d_cols * 3];

	for(int idx=0; idx<d_rows; idx++){
		for(int jdx=0; jdx<d_cols; jdx++){
			for(int kdx=0; kdx<3; kdx++){
				image_file>>input_img[idx*d_cols*3 + jdx*3 + kdx];
			}
		}
	}
	image_file.close();

	int q_rows,q_cols;
	query_file>>q_rows;
	query_file>>q_cols;

	int *query_img = new int[q_rows * q_cols * 3];
	
	for(int idx=0;idx<q_rows;idx++){
		for(int jdx=0;jdx<q_cols;jdx++){
			for(int kdx=0;kdx<3;kdx++){
				query_file>>query_img[idx*q_cols*3 + jdx*3 + kdx];
			}
		}
	}

	query_file.close();

	//memory allocation for gpu
	int *dM, *dQ;
	hipMalloc(&dM, d_rows*d_cols * 3 * sizeof(int));
	hipMalloc(&dQ, q_rows*q_cols * 3 * sizeof(int));
	hipMemcpy(dM, input_img, d_rows * d_cols * 3 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dQ, query_img, q_rows * q_cols * 3 * sizeof(int), hipMemcpyHostToDevice);

	//get query imae avg
	int qavg = 0;
	for(int i=0; i<q_rows * q_cols * 3; i+=3){
		qavg += (query_img[i] + query_img[i+1] + query_img[i+2])/3;
	}
	qavg /= (q_cols * q_rows);
	// cout<<"qavg :"<<qavg<<'\n';

	int th1 = 1;

	//kernel invocation
	int N = (d_rows - q_rows + 1) * (d_cols - q_cols + 1);
	AverageFinder<<<(N + 255)/256,256>>>(dM, dQ, d_rows, d_cols, q_rows, q_cols, qavg, th1);


	hipDeviceSynchronize();
	//getback from the kernel
	int *Topn = new int[topn * 3];

	hipFree(dM);
	hipFree(dQ);
	//calculate topntriplets

	ofstream output_file("output.txt", ios::out);
	for(int i=0;i<topn * 3;i+=3){
		output_file<<Topn[i]<<" "<<Topn[i+1]<<" "<<Topn[i+2]<<"\n";
	}
	output_file.close();
	return 0;
}
