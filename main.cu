
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <map>
#include <bits/stdc++.h>
#include <math.h>
#include <queue>
#include <vector>
#include<string>

#define PI 3.141592 

using namespace std;

__device__
double distbtw(double a, double b, double c, double d)
{
	double x = (a-c) * (a-c);
	double y = (b-d) * (b-d);
	return sqrt(x + y);
}
__global__ 
void AverageFinder(int* dM, int *dQ, double *dR, int d_rows, int d_cols, int q_rows, int q_cols, int qavg, int th1, int angle)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	double y = i / d_cols;
	double x = i % d_cols;
	double leftmost, rightmost, topmost, bottommost;
	double sqrt2 = sqrt(2.0f);
	if (angle == 1){
		leftmost = x - ((q_rows - 1)  / sqrt2);
		rightmost = x + ((q_cols - 1) / sqrt2);
		topmost = y + ((q_cols - 1) / sqrt2) + ((q_rows - 1) / sqrt2);
		bottommost = y;
	}

	else if(angle == 0){
		leftmost = x;
		rightmost = x + q_cols - 1;
		topmost = y + q_rows - 1;
		bottommost = y;
	}

	else if(angle == -1){
		rightmost = x + ((q_cols - 1) / sqrt2) + ((q_rows - 1)/ sqrt2);
		leftmost = x;
		topmost = y + ((q_rows - 1) / sqrt2);
		bottommost = y - ((q_cols - 1) / sqrt2);
	}
	 
	// printf("topmost:%f, bottommost:%f, leftmost:%f, rightmost:%f\n", topmost, bottommost, leftmost, rightmost);
	if(topmost >= d_rows || bottommost < 0 || leftmost < 0 || rightmost >= d_cols){
		dR[i] = -1.0f;
		return;
	}

	int avg = 0;
	for(int r = bottommost; r <= topmost; r++){
		for(int c = leftmost; c <= rightmost; c++){
			int pavg = 0;
			int point = r * d_cols + c;
			for(int k = 0; k < 3; k++){
				pavg += dM[point * 3 + k];
			}
			avg += pavg/3;
		}
	}
	avg /= ((topmost - bottommost + 1) * (rightmost - leftmost + 1));
	// printf("threadidx:%d avg:%d\n",i, avg);
	if(abs(qavg - avg) <= th1){
		double total = 0;
		for(int r = 0; r<q_rows; r++){
			for(int c = 0; c<q_cols; c++){
				double baseang;
				if(angle == 1)
					baseang = 45 * PI / 180;
				else if(angle == -1)
					baseang = -45 * PI / 180;
				else if(angle == 0)
					baseang = 0;
				double d = distbtw(r + x, c + y, x, y);
				double ang = baseang + atan2((double)r, (double)c);
				double rx = x + cos(ang) * d;
				double ry = y + sin(ang) * d;
				double ceilrx = ceil(rx), floorrx = floor(rx), ceilry = ceil(ry), floorry = floor(ry);
				double colorR, colorG, colorB;
				if(((ceilrx - rx) > 1e-10 && (rx - floorrx) > 1e-10)|| ((ry - floorry) > 1e-10 && (ceilry - ry) > 1e-10)){
					//bilinear interpolation
					// printf("%d doing bilinear interpolation, baseang%f ang%f d:%f ceilrx:%f floorrx:%f rx:%f ceilry:%f floorry:%f ry:%f\n", i, baseang, ang, d, ceilrx, floorrx, rx, ceilry, floorry, ry);
					colorR = dM[(int)(floorry * d_cols + floorrx)*3]*(ceilrx - rx)*(ceilry - ry) + dM[(int)(floorry * d_cols + ceilrx)*3]*(rx - floorrx)*(ceilry - ry) + dM[(int)(ceilry * d_cols + floorrx)*3]*(ceilrx - rx)*(ry - floorry) + dM[(int)(ceilry * d_cols + ceilrx)*3]*(rx - floorrx)*(ry - floorry);
					colorG = dM[(int)(1 + (floorry * d_cols + floorrx)*3)]*(ceilrx - rx)*(ceilry - ry) + dM[(int)(1 + (floorry * d_cols + ceilrx)*3)]*(rx - floorrx)*(ceilry - ry) + dM[(int)(1 + (ceilry * d_cols + floorrx)*3)]*(ceilrx - rx)*(ry - floorry) + dM[(int)(1 + (ceilry * d_cols + ceilrx)*3)]*(rx - floorrx)*(ry - floorry);
					colorB = dM[(int)(2 + (floorry * d_cols + floorrx)*3)]*(ceilrx - rx)*(ceilry - ry) + dM[(int)(2 + (floorry * d_cols + ceilrx)*3)]*(rx - floorrx)*(ceilry - ry) + dM[(int)(2 + (ceilry * d_cols + floorrx)*3)]*(ceilrx - rx)*(ry - floorry) + dM[(int)(2 + (ceilry * d_cols + ceilrx)*3)]*(rx - floorrx)*(ry - floorry);

				}
				else{
					// printf("%d doing normal interpolation, baseang%f ang%f ceilrx:%f floorrx:%f rx:%f ceilry:%f floorry:%f ry:%f\n", i, baseang, ang, ceilrx, floorrx, rx, ceilry, floorry, ry);
					colorR = dM[(int)(ry * d_cols + rx)*3];
					colorG = dM[(int)(1 + (ry * d_cols + rx)*3)];
					colorB = dM[(int)(2 + (ry * d_cols + rx)*3)];
				}
				double diffR = colorR - dQ[(r * q_cols + c)*3];
				double diffG = colorG - dQ[1 + (r * q_cols + c)*3];
				double diffB = colorB - dQ[2 + (r * q_cols + c)*3];
				total += (diffR*diffR + diffG*diffG + diffB*diffB);
			}
		}
		total /= (q_cols*q_rows*3);
		total = sqrt(total);
		dR[i] = total;
		//printf("%d (%f,%f) with avg:%d is close, RMSD:%f\n",i,x,y,avg,total);
	}
	else{
		// printf("%d (%f,%f) with avg:%d is not close\n",i,x,y,avg);
		dR[i] = -1.0f;
	}
}

void calcTopn(priority_queue<pair<double, vector<int> > > &Topn, double *dR, int N, int topn, int angle,int thresh){
	for(int i=0;i<N;i++){
		if(Topn.size() >= topn && dR[i]>=0 && dR[i]<=thresh){
			//cout<<"got:"<<i<<" "<<dR[i]<<" "<<angle<<"\n";
			pair<double, vector<int> > topele = Topn.top();
			if(topele.first > dR[i]){
				Topn.pop();
				vector<int> temp;
				temp.push_back(i);
				temp.push_back(angle);
				Topn.push(make_pair(dR[i], temp));
			}
		}
		else if(dR[i]>=0 && dR[i]<=thresh){
			//cout<<"got:"<<i<<" "<<dR[i]<<" "<<angle<<"\n";
			vector<int> temp;
			temp.push_back(i);
			temp.push_back(angle);
			Topn.push(make_pair(dR[i], temp));
		}
	}
}

int main(int argc, char* argv[]){
	if(argc < 6){
		cout<<"insufficient args provided\n";
		return -1;
	}

	ifstream image_file(argv[1], ios::in);
	ifstream query_file(argv[2], ios::in);
	int threshold1 = atoi(argv[4]); 		// for summation filtering..
	int threshold2 = atoi(argv[3]);			// for rdmsa
	int topn = atoi(argv[5]);

	int d_rows,d_cols;
	image_file>>d_rows;
	image_file>>d_cols;

	//cerr<<"break1\n";

	int *input_img = new int[d_rows * d_cols * 3];

	for(int idx=d_rows-1; idx>=0; idx--){
		for(int jdx=0; jdx<d_cols; jdx++){
			for(int kdx=0; kdx<3; kdx++){
				image_file>>input_img[idx*d_cols*3 + jdx*3 + kdx];
			}
		}
	}
	image_file.close();

	int q_rows,q_cols;
	query_file>>q_rows;
	query_file>>q_cols;

	//cerr<<"break2\n";
	int *query_img = new int[q_rows * q_cols * 3];
	
	for(int idx=q_rows-1;idx>=0;idx--){
		for(int jdx=0;jdx<q_cols;jdx++){
			for(int kdx=0;kdx<3;kdx++){
				query_file>>query_img[idx*q_cols*3 + jdx*3 + kdx];
			}
		}
	}

	query_file.close();

	//memory allocation for gpu
	int *dM, *dQ;
	hipMalloc(&dM, d_rows*d_cols * 3 * sizeof(int));
	hipMalloc(&dQ, q_rows*q_cols * 3 * sizeof(int));
	hipMemcpy(dM, input_img, d_rows * d_cols * 3 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dQ, query_img, q_rows * q_cols * 3 * sizeof(int), hipMemcpyHostToDevice);

	//get query imae avg
	// int N = (d_rows - q_rows + 1) * (d_cols - q_cols + 1);
	int N = d_rows * d_cols;
	int qavg = 0;
	int th1 = threshold1;
	for(int i=0; i<q_rows * q_cols * 3; i+=3){
		qavg += (query_img[i] + query_img[i+1] + query_img[i+2])/3;
	}
	qavg /= (q_cols * q_rows);
	//cout<<"qavg :"<<qavg<<'\n';
	//cout<<"threshold1:"<<th1<<" threshold2:"<<threshold2<<" topn:"<<topn<<"\n";


	priority_queue<pair<double, vector<int> > > Topn;
	
	double *dR;
	double *R = new double[N];
	hipMalloc(&dR, N * sizeof(double));

	//kernel invocation
	AverageFinder<<<(N + 255)/256,256>>>(dM, dQ, dR, d_rows, d_cols, q_rows, q_cols, qavg, th1, 0);
	hipDeviceSynchronize();
	hipMemcpy(R, dR, N * sizeof(double), hipMemcpyDeviceToHost);
	calcTopn(Topn,R,N,topn,0,threshold2);

	AverageFinder<<<(N + 255)/256,256>>>(dM, dQ, dR, d_rows, d_cols, q_rows, q_cols, qavg, th1, 1);
	hipDeviceSynchronize();
	hipMemcpy(R, dR, N * sizeof(double), hipMemcpyDeviceToHost);
	calcTopn(Topn,R,N,topn,1,threshold2);

	AverageFinder<<<(N + 255)/256,256>>>(dM, dQ, dR, d_rows, d_cols, q_rows, q_cols, qavg, th1, -1);
	hipDeviceSynchronize();
	hipMemcpy(R, dR, N * sizeof(double), hipMemcpyDeviceToHost);			// Sidharth: optimize this
	calcTopn(Topn,R,N,topn,-1,threshold2);

	hipFree(dR);
	hipFree(dM);
	hipFree(dQ);

	
	vector<vector<int> > ans;
	while(Topn.size()>0)
	{
		ans.push_back(Topn.top().second);
		//cerr<<Topn.top().first<<"\n";
		Topn.pop();
	}

	ofstream output_file("output.txt", ios::out);

	for(int idx=ans.size()-1;idx>=0;idx--)
	{
		output_file << ans[idx][0]/d_cols;
		output_file << " ";
		output_file << ans[idx][0]%d_cols;
		output_file << " ";

		if(int(ans[idx][1])==1) output_file << "45";
		else if(int(ans[idx][1])==-1) output_file << "-45";
		else output_file << "0";

		output_file << "\n";
	}
	output_file.close();
	return 0;
}