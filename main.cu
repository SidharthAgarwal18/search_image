
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <map>
#include <bits/stdc++.h>
#include <math.h>

using namespace std;
__global__ 
void AverageFinder(int* dM, int *dQ, double *dR, int d_rows, int d_cols, int q_rows, int q_cols, int qavg, int th1, int angle)
{
	int avg = 0;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	double x = i / d_cols;
	double y = i % d_cols;
	double leftmost, rightmost, topmost, bottommost;
	double sqrt2 = sqrt(2.0f);
	if (angle == 1){
		leftmost = x - (q_rows / sqrt2);
		rightmost = x + (q_cols / sqrt2);
		topmost = y + (q_cols / sqrt2) + (q_rows / sqrt2);
		bottommost = y;
	}

	else if(angle == 0){
		leftmost = x;
		rightmost = x + q_cols;
		topmost = y + q_rows;
		bottommost = y;
	}

	else if(angle == -1){
		rightmost = x + (q_cols / sqrt2) + (q_rows / sqrt2);
		leftmost = x;
		topmost = y + (q_rows / sqrt2);
		bottommost = y - (q_cols / sqrt2);
	}
	 
	printf("topmost:%f, bottommost:%f, leftmost:%f, rightmost:%f\n", topmost, bottommost, leftmost, rightmost);
	if(topmost >= d_rows || bottommost < 0 || leftmost < 0 || rightmost >= d_cols){
		dR[i] = -1.0f;
		return;
	}


	// printf("threadidx:%d\n",i);
	//add check to remove all for(int r = 0; r<q_rows; r++){
	// 	for(int c = 0; c<q_cols; c++){
	// 		int point = i + r*d_cols + c;
	// 		int pavg = 0;
	// 		for(int k = 0; k<3; k++){
	// 			pavg += dM[point * 3 + k];
	// 		}
	// 		avg += pavg / 3;
	// 	}
	// }

	// avg /= (q_rows * q_cols);
	// //printf("avg : %d\n",avg);
	// if(abs(qavg - avg) <= th1){
	// 	double total = 0;
	// 	for(int r = 0; r<q_rows; r++){
	// 		for(int c = 0; c<q_cols; c++){
	// 			for(int k = 0; k<3; k++){
	// 				long v = dM[i*3 + r*d_cols*3 + c*3 + k] - dQ[r*q_cols*3 + c*3 + k];
	// 				total += v * v;
	// 			}
	// 		}
	// 	}
	// 	total /= (q_cols*q_rows*3);
	// 	total = sqrt(total);
	// 	dR[i] = total;
	// 	printf("%d is close, RMSD : %f\n",i,total);
	// }
	// else{
	// 	dR[i] = -1.0f;
	// }overlaps that are outside data_image for all angles
	// for(int r = 0; r<q_rows; r++){
	// 	for(int c = 0; c<q_cols; c++){
	// 		int point = i + r*d_cols + c;
	// 		int pavg = 0;
	// 		for(int k = 0; k<3; k++){
	// 			pavg += dM[point * 3 + k];
	// 		}
	// 		avg += pavg / 3;
	// 	}
	// }

	// avg /= (q_rows * q_cols);
	// //printf("avg : %d\n",avg);
	// if(abs(qavg - avg) <= th1){
	// 	double total = 0;
	// 	for(int r = 0; r<q_rows; r++){
	// 		for(int c = 0; c<q_cols; c++){
	// 			for(int k = 0; k<3; k++){
	// 				long v = dM[i*3 + r*d_cols*3 + c*3 + k] - dQ[r*q_cols*3 + c*3 + k];
	// 				total += v * v;
	// 			}
	// 		}
	// 	}
	// 	total /= (q_cols*q_rows*3);
	// 	total = sqrt(total);
	// 	dR[i] = total;
	// 	printf("%d is close, RMSD : %f\n",i,total);
	// }
	// else{
	// 	dR[i] = -1.0f;
	// }

}

int main(int argc, char* argv[]){
	if(argc < 5){
		cout<<"insufficient args provided\n";
		return -1;
	}

	ifstream image_file(argv[1], ios::in);
	ifstream query_file(argv[2], ios::in);
	int threshold = atoi(argv[3]);
	int topn = atoi(argv[4]);

	int d_rows,d_cols;
	image_file>>d_rows;
	image_file>>d_cols;

	int *input_img = new int[d_rows * d_cols * 3];

	for(int idx=0; idx<d_rows; idx++){
		for(int jdx=0; jdx<d_cols; jdx++){
			for(int kdx=0; kdx<3; kdx++){
				image_file>>input_img[idx*d_cols*3 + jdx*3 + kdx];
			}
		}
	}
	image_file.close();

	int q_rows,q_cols;
	query_file>>q_rows;
	query_file>>q_cols;

	int *query_img = new int[q_rows * q_cols * 3];
	
	for(int idx=0;idx<q_rows;idx++){
		for(int jdx=0;jdx<q_cols;jdx++){
			for(int kdx=0;kdx<3;kdx++){
				query_file>>query_img[idx*q_cols*3 + jdx*3 + kdx];
			}
		}
	}

	query_file.close();

	//memory allocation for gpu
	int *dM, *dQ;
	hipMalloc(&dM, d_rows*d_cols * 3 * sizeof(int));
	hipMalloc(&dQ, q_rows*q_cols * 3 * sizeof(int));
	hipMemcpy(dM, input_img, d_rows * d_cols * 3 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dQ, query_img, q_rows * q_cols * 3 * sizeof(int), hipMemcpyHostToDevice);

	//get query imae avg
	// int N = (d_rows - q_rows + 1) * (d_cols - q_cols + 1);
	int N = d_rows * d_cols;
	int qavg = 0;
	for(int i=0; i<q_rows * q_cols * 3; i+=3){
		qavg += (query_img[i] + query_img[i+1] + query_img[i+2])/3;
	}
	qavg /= (q_cols * q_rows);
	// cout<<"qavg :"<<qavg<<'\n';

	int th1 = 1;
	
	double *dR;
	hipMalloc(&dR, N * sizeof(double));

	//kernel invocation
	AverageFinder<<<(N + 255)/256,256>>>(dM, dQ, dR, d_rows, d_cols, q_rows, q_cols, qavg, th1, 0);
	hipDeviceSynchronize();
	AverageFinder<<<(N + 255)/256,256>>>(dM, dQ, dR, d_rows, d_cols, q_rows, q_cols, qavg, th1, 1);
	hipDeviceSynchronize();
	AverageFinder<<<(N + 255)/256,256>>>(dM, dQ, dR, d_rows, d_cols, q_rows, q_cols, qavg, th1, -1);
	hipDeviceSynchronize();
	//getback from the kernel
	map<int, double> Topn;
	double *R = new double[N];
	hipMemcpy(R, dR, N * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(dR);
	hipFree(dM);
	hipFree(dQ);

	//calculate topntriplets
	// for(int i=0; i<N; i++){
	// 	if(Topn.size() < topn){
	// 		Topn.emplace_back(R[i], i);
	// 		sort(Topn.begin(), Topn.end());
	// 	}
	// 	else{
	// 		if(Topn[topn-1].first > R[i]){
	// 			Topn.emplace_back(R[i], i);
	// 			sort(Topn.begin(), Topn.end());
	// 			Topn.pop_back();
	// 		}
	// 	}
	// }


	ofstream output_file("output.txt", ios::out);
	output_file.close();
	return 0;
}
