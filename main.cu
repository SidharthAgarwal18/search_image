
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <string>
#include <cstdio>
#include <bits/stdc++.h>
#include <vector>
#include <stdlib.h>
#include <utility>

using namespace std;

__global__ void AverageFinder(int* dM, int* dA, int d_rows, int d_cols, int q_rows, int q_cols){
	int avg = 100;
	printf("Thread: %d%d\n", threadIdx.x, threadIdx.y);
	for(int i=threadIdx.x; i<threadIdx.x + q_rows; i++){
		for(int j=threadIdx.y; i<threadIdx.y + q_cols; j++){
			avg += (dM[i*q_cols*3 + j*3] + dM[i*q_cols*3 + j*3 + 1] + dM[i*q_cols*3 + j*3 + 2]);
		}
	}
	printf("sum = %d\n", avg);
	// avg /= q_rows * q_cols;
	printf("avg = %d\n", avg);
	dA[threadIdx.x * (d_rows - q_rows + 1) + threadIdx.y] = avg;
}

int main(int argc, char* argv[]){

	if(argc < 5){
		cout<<"insufficient args provided\n";
		return -1;
	}

	ifstream image_file(argv[1], ios::in);
	ifstream query_file(argv[2], ios::in);
	int threshold = atoi(argv[3]);
	int topn = atoi(argv[4]);

	int d_rows,d_cols;
	image_file>>d_rows;
	image_file>>d_cols;

	int *input_img = new int[d_rows * d_cols * 3];

	for(int idx=0; idx<d_rows; idx++){
		for(int jdx=0; jdx<d_cols; jdx++){
			for(int kdx=0; kdx<3; kdx++){
				image_file>>input_img[idx*d_cols*3 + jdx*3 + kdx];
			}
		}
	}
	image_file.close();

	int q_rows,q_cols;
	query_file>>q_rows;
	query_file>>q_cols;

	int *query_img = new int[q_rows * q_cols * 3];
	
	for(int idx=0;idx<q_rows;idx++){
		for(int jdx=0;jdx<q_cols;jdx++){
			for(int kdx=0;kdx<3;kdx++){
				query_file>>query_img[idx*q_cols*3 + jdx*3 + kdx];
			}
		}
	}

	query_file.close();

	//memory allocation for gpu
	int *dM, *dQ;
	hipMalloc(&dM, d_rows*d_cols * 3 * sizeof(int));
	hipMalloc(&dQ, q_rows*q_cols * 3 * sizeof(int));
	hipMemcpy(dM, input_img, d_rows * d_cols * 3 * sizeof(int), hipMemcpyDefault);
	hipMemcpy(dQ, query_img, q_rows * q_cols * 3 * sizeof(int), hipMemcpyDefault);

	//storing average distances
	int *dA;				
	hipMalloc(&dA, (d_rows - q_rows + 1) * (d_cols - q_cols + 1) * sizeof(int));

	//kernel invocation
	dim3 dimBlock((d_rows - q_rows + 1) , (d_cols - q_cols + 1));
	AverageFinder<<<1, dimBlock>>>(dM, dA, d_rows, d_cols, q_rows, q_cols);


	hipDeviceSynchronize();
	//getback from the kernel
	int *A = new int[(d_rows - q_rows + 1) * (d_cols - q_cols + 1)];
	int *Topn = new int[topn * 3];
	hipMemcpy(A, dA, (d_rows - q_rows + 1) * (d_cols - q_cols + 1) * sizeof(int), hipMemcpyDefault);
	hipFree(dM);
	hipFree(dQ);
	hipFree(dA);

	//debug
	for(int i=0;i< (d_rows - q_rows + 1) * (d_cols - q_cols + 1); i++){
		cout<<A[i]<<" ";
	}
	cout<<"\nDone printing A\n";
	
	//calculate topntriplets

	ofstream output_file("output.txt", ios::out);
	for(int i=0;i<topn * 3;i+=3){
		output_file<<Topn[i]<<" "<<Topn[i+1]<<" "<<Topn[i+2]<<"\n";
	}
	output_file.close();
	return 0;
}
